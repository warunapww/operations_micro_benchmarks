
#include <papi.h>
#include <nvml.h>
#include <stdio.h>
//#include <time.h>
#include <unistd.h>

#include "high_resolution_power.h"

int sleep_time_after_kernel_call = 60; //s
int reps = 1; //getenv
unsigned int device_id = 0;
long long time_kernel_start_ex = 0;

long long kernel_execution_time = 0; //nano seconds

nvmlReturn_t nvml_result;
nvmlDevice_t nvml_device;

void handle_papi_error(int retval)
{
	PAPI_perror((char *) "Fehler");
	printf((char *) "PAPI error %d: %s\n", retval, PAPI_strerror(retval));
//	exit(1);
}

int nvml_finalize(nvmlReturn_t result) {
  result = nvmlShutdown();
  if (NVML_SUCCESS != result) {
    printf("Failed to shutdown NVML: %s\n", nvmlErrorString(result));
    return -1;
  }
  return 1;
}

unsigned int synchronizeTime(long long & cpu_time, nvmlDevice_t nvml_device, unsigned int &temperature, int &pstate)
{
	unsigned int gpu_value = 0;
	unsigned int last_gpu_value = 0;


  int device_id = 0;

	nvmlReturn_t nvml_result;
	nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
	if (nvml_result != NVML_SUCCESS) printf("NVML error: %s.\n", nvmlErrorString(nvml_result));
	last_gpu_value = gpu_value;

	while (gpu_value == last_gpu_value)
	{
		last_gpu_value = gpu_value;
		nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
		if (nvml_result != NVML_SUCCESS) printf("NVML error: %d.\n", nvmlErrorString(nvml_result));
	}

	cpu_time = PAPI_get_real_nsec();
  
///////////////////////////////////////
  nvml_result = nvmlDeviceGetTemperature(nvml_device, NVML_TEMPERATURE_GPU, &temperature);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get temperature of the device %i: %s\n", device_id, nvmlErrorString(nvml_result));
   // nvml_finalize(nvml_result);
   // return -1;
  }
/*
  nvml_result = nvmlDeviceGetFanSpeed(nvml_device, fan_usage);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get fan speed of the device %i: %s\n", device_id, nvmlErrorString(nvml_result));
   // nvml_finalize(nvml_result);
   // return -1;
  }
  */
  nvmlPstates_t p_state;
  nvml_result = nvmlDeviceGetPerformanceState(nvml_device, &p_state);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get perf state of the device %i: %s\n", device_id, nvmlErrorString(nvml_result));
   // nvml_finalize(nvml_result);
   // return -1;
  }

  pstate = p_state;
//////////////////////////////////////

	return gpu_value;
}


/*unsigned int synchronizeTime(long long & cpu_time, nvmlDevice_t nvml_device)
{
	unsigned int gpu_value = 0;
	unsigned int last_gpu_value = 0;
	nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
	if (nvml_result != NVML_SUCCESS) printf("NVML error: %s.\n", nvmlErrorString(nvml_result));
	last_gpu_value = gpu_value;

	while (gpu_value == last_gpu_value)
	{
		last_gpu_value = gpu_value;
		nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_value);
		if (nvml_result != NVML_SUCCESS) printf("NVML error: %d.\n", nvmlErrorString(nvml_result));
	}

	cpu_time = PAPI_get_real_nsec();

	return gpu_value;
}*/

int nvml_init() { 
  char *nvml_reps = getenv("NVML_REPS");
  if (nvml_reps != NULL) {
    reps = atoi(nvml_reps);
  }

  char *nvml_sleep_time = getenv("NVML_SLEEP_TIME");
  if (nvml_sleep_time != NULL) {
    sleep_time_after_kernel_call = atoi(nvml_sleep_time);
  }

	// Initialize the PAPI library
	int retval = PAPI_NULL;
  retval = PAPI_library_init(PAPI_VER_CURRENT);
	if (retval != PAPI_VER_CURRENT) {
		printf("PAPI library init error: %d!\n", retval);
    handle_papi_error(retval);
		return -1;
	} 

  //initializing nvml
  nvml_result = nvmlInit();
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to initialize NVML: %s\n", nvmlErrorString(nvml_result));
    return -1;
  }

	nvml_result = nvmlDeviceGetHandleByIndex(device_id, &nvml_device);
  if (NVML_SUCCESS != nvml_result)
  {
    printf("Failed to get handle for device %i: %s\n", device_id, nvmlErrorString(nvml_result));
    nvml_result = nvmlShutdown();
    nvml_finalize(nvml_result);
    return -1;
  }
  
  printf("#NUM_REPS: %d COOLDOWN_TIME: %ds\n", reps, sleep_time_after_kernel_call);

  return 0;
}

/*void set_kernel_execution_time(long long time) {
  kernel_execution_time = time;
}*/

int power_profile(void (*call_cuda_kernel)(void)) {
  struct timespec sleep_time;
  sleep_time.tv_sec = sleep_time_after_kernel_call;
  sleep_time.tv_nsec = 0;

  // perform actual energy measurement
	for (int n_ex = 0; n_ex < reps; n_ex++)
	{
		// wait a random time
		sleep_time.tv_nsec = rand()%(DELTA_T) + DELTA_T;
		nanosleep(&sleep_time, NULL);

		// call GPU kernel
		long long time_start_kernel = PAPI_get_real_nsec();
		call_cuda_kernel();
		long long time_current = time_start_kernel;

		unsigned int gpu_power_before;
		nvml_result = nvmlDeviceGetPowerUsage(nvml_device, &gpu_power_before);
		if (nvml_result != NVML_SUCCESS) {
      printf("NVML error: %s.\n", nvmlErrorString(nvml_result));
    }

		bool gpu_power_before_printed = false; // indicates wether the first power measurement (before the start of the GPU kernel) has already been printed out

		// continually retrieve power values of the GPU
		while (time_current < time_start_kernel + kernel_execution_time)
		{
			long long time_running_update;
      unsigned int temperature;
      int pstate;
			unsigned int gpu_power = synchronizeTime(time_running_update, nvml_device, temperature, pstate);
			//unsigned int gpu_power = synchronizeTime(time_running_update, nvml_device);

			if (!gpu_power_before_printed)
			{
				printf("%.5f ms\t%.5f W\t%u C\t%d\t%.5f ms\n", (time_running_update - time_start_kernel - DELTA_T) / 1e6, gpu_power_before / 1e3, temperature, pstate, (time_current - time_kernel_start_ex - DELTA_T) / 1e6);
				gpu_power_before_printed = true;
			}

			printf("%.5f ms\t%.5f W\t%u C\t%d\t%.5f ms\n", (time_running_update - time_start_kernel) / 1e6, gpu_power / 1e3, temperature, pstate, (time_current - time_kernel_start_ex) / 1e6);
			time_current = time_running_update;
		}

		hipDeviceSynchronize();

    for (int tt=0; tt<10; tt++) {
      usleep(20000);
      long long tru;
      unsigned int temperature;
      int pstate;
      unsigned int gp = synchronizeTime(tru, nvml_device, temperature, pstate);
      printf("%.5f ms\t%.5f W\t%u C\t%d\t%.5f ms\n", (tru - time_start_kernel) / 1e6, gp / 1e3, temperature, pstate, (time_current - time_kernel_start_ex) / 1e6); 
			time_current = tru;
    }

// 		long long time_finish_kernel = PAPI_get_real_nsec();
	}

	long long time_simulation_end;
  unsigned int t;
  int p;
	synchronizeTime(time_simulation_end, nvml_device, t, p);
	printf("# end time: %.5f\n", (time_simulation_end - time_kernel_start_ex) / 1e6);

  return 0;
}


int high_resolution_power_profile(void (*call_cuda_kernel)(void)) {
  int error = nvml_init();
  if (error != 0) {
    return error;
  }
//---------------------------------------------------------------------------------------------------------------------------------
  //set kernel execution time
  call_cuda_kernel();
  call_cuda_kernel();
  hipDeviceSynchronize();
  time_kernel_start_ex = PAPI_get_real_nsec();
  call_cuda_kernel();
  hipDeviceSynchronize();
  long long time_kernel_finish_ex = PAPI_get_real_nsec();
  kernel_execution_time = time_kernel_finish_ex - time_kernel_start_ex; 

  printf("#Kernel execution time: %.5fms\n", kernel_execution_time/1e6);
//---------------------------------------------------------------------------------------------------------------------------------
  return power_profile(call_cuda_kernel);  
}

// in nanoseconds
long long get_exec_time_in_nanoseconds(void (*call_cuda_kernel)(void)) {
   //get kernel execution time
  call_cuda_kernel();
  call_cuda_kernel();
  hipDeviceSynchronize();
  time_kernel_start_ex = PAPI_get_real_nsec();
  call_cuda_kernel();
  hipDeviceSynchronize();
  long long time_kernel_finish_ex = PAPI_get_real_nsec();
  kernel_execution_time = time_kernel_finish_ex - time_kernel_start_ex;

  //printf("#Kernel execution time: %.5fms\n", kernel_execution_time/1e6);

  return kernel_execution_time;

}
