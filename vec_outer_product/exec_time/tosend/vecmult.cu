#include "hip/hip_runtime.h"
///
/// vecadd.cu
/// For CSU CS575 Spring 2011
/// Instructor: Wim Bohm
/// Based on code from the CUDA Programming Guide
/// Modified by Wim Bohm and David Newman
/// Created: 2011-02-03
/// Last Modified: 2011-03-03 DVN
///
/// Add two Vectors A and B in C on GPU using
/// a kernel defined according to vecAddKernel.h
/// Students must not modify this file. The GTA
/// will grade your submission using an unmodified
/// copy of this file.
/// 

// Includes
#include <stdio.h>
#include <cutil.h>
#include "vecmultKernel.h"

// Defines
//#define GridWidth 60
//#define BlockWidth 128

// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* h_C; 
float* d_A; 
float* d_B; 
float* d_C; 

// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int ValuesPerThread; // number of values per thread
    int N; //Vector size
	int k; // no. of repeatitions
	int gridWidth = 60;
	int blockWidth = 1;

	// Parse arguments.
    if(argc != 5){
     printf("Usage: %s ValuesPerThread Iterations\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     printf("Iterations is the number of repeatitions done by each thread.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
      sscanf(argv[2], "%d", &k);
      sscanf(argv[3], "%d", &gridWidth);
      sscanf(argv[4], "%d", &blockWidth);
    }      

	int size_A = blockWidth * ValuesPerThread;
	int size_B = gridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);

    // Determine the number of threads .
    // N is the total number of values to be in a vector
//    N = ValuesPerThread * gridWidth * blockWidth;
//    printf("Total vector size: %d : Iterations: %d\n", N, k); 
    // size_t is the total number of bytes for a vector.
//    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.
    dim3 dimGrid(gridWidth);                    
    dim3 dimBlock(blockWidth);                 

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size_A*sizeof(float));
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size_B*sizeof(float));
    if (h_B == 0) Cleanup(false);
    h_C = (float*)malloc(size_A*sizeof(float)*size_B*sizeof(float));
    if (h_C == 0) Cleanup(false);
	
//	printf("1 \n");

    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size_A*sizeof(float));
    if (error != hipSuccess) Cleanup(false);
	//printf("1.1\n");
    error = hipMalloc((void**)&d_B, size_B*sizeof(float));
    if (error != hipSuccess) Cleanup(false);
//	printf("1.2\n");
    error = hipMalloc((void**)&d_C, size_A*sizeof(float)*size_B*sizeof(float));
    if (error != hipSuccess) Cleanup(false);

//	printf("2\n");
    // Initialize host vectors h_A and h_B
    int i, j;
    for(i=0; i <size_A; ++i){
     h_A[i] = (float)i;
    }
    for(i=0; i <size_B; ++i){
     h_B[i] = (float)(N-i);   
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size_A*sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size_B*sizeof(float), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);

//	printf("3\n");
    // Warm up
    MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    hipDeviceSynchronize();

//	printf("4\n");
    // Initialize timer
    unsigned int timer = 0;
    cutCreateTimer(&timer);
    cutStartTimer(timer);

    // Invoke kernel
    MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}

//	printf("5\n");
	// Compute elapsed time 
    hipDeviceSynchronize();
    cutStopTimer(timer);
    float time = cutGetTimerValue(timer);

	// Compute floating point operations per second.
    //double nFlops = size_A*size_B*k*2 + 2*k*ValuesPerThread*gridWidth*blockWidth;
    //double nFlops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)2*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
    double nFlops = (double)size_A*(double)size_B*(double)k*(double)2;
    float nFlopsPerSec = 1e3*nFlops/time;
    float nGFlopsPerSec = nFlopsPerSec*1e-9;
	//printf("%f :: %f", (double)size_A*(double)size_B*(double)k*(double)2, (double)2*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth);
	// Compute transfer rates.
    float nBytes = size_A*sizeof(float) + size_B*sizeof(float) + size_A*sizeof(float)*size_B*sizeof(float); // 2N words in, N*N word out
    float nBytesPerSec = 1e3*nBytes/time;
    float nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    printf( "Time: %f (ms), GFLOPS: %f, GBytesS: %f\n", 
             time, nGFlopsPerSec, nGBytesPerSec);
     
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size_A*sizeof(float)*size_B*sizeof(float), hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
			float val = h_C[i*size_B+j];
			if (fabs(val - h_A[i]*h_B[j]) > 1e-5) {
				printf("Result error: i=%d, j=%d, expected %f, got %f\n", i, j, h_A[i]*h_B[j], val);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    }
    printf("Test %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	// Clean up and exit.
    cutDeleteTimer( timer);
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
   checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


