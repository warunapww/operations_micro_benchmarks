#include "hip/hip_runtime.h"
/** x = 3
**/    #include "vecmultKernel.h"
__global__ void MultiplyVectors(const float* A, const float* B, float* C)
{
	int B_start_index = blockIdx.x*ValuesPerThread;
	int A_start_index = threadIdx.x*ValuesPerThread;
	int C_width = gridDim.x*ValuesPerThread;


	int t;
	float c_0_0, c_0_1, c_0_2, c_1_0, c_1_1, c_1_2, c_2_0, c_2_1, c_2_2;
	float a_0, a_1, a_2;
	float b_0, b_1, b_2;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];
	a_2 = A[A_start_index+2];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];
	b_2 = B[B_start_index+2];


	c_0_0 = 0;
	c_0_1 = 0;
	c_0_2 = 0;
	c_1_0 = 0;
	c_1_1 = 0;
	c_1_2 = 0;
	c_2_0 = 0;
	c_2_1 = 0;
	c_2_2 = 0;


	for (t = 0; t < k; t++) {
		c_0_0 += a_0*b_0;
		c_0_1 += a_0*b_1;
		c_0_2 += a_0*b_2;
		c_1_0 += a_1*b_0;
		c_1_1 += a_1*b_1;
		c_1_2 += a_1*b_2;
		c_2_0 += a_2*b_0;
		c_2_1 += a_2*b_1;
		c_2_2 += a_2*b_2;


		a_0 = a_0*1.1f+1.7f;
		a_1 = a_1*1.1f+1.7f;
		a_2 = a_2*1.1f+1.7f;


		b_0 = b_0*1.1f+1.7f;
		b_1 = b_1*1.1f+1.7f;
		b_2 = b_2*1.1f+1.7f;


	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c_0_0;
	C[(A_start_index+0)*C_width + B_start_index+1] = c_0_1;
	C[(A_start_index+0)*C_width + B_start_index+2] = c_0_2;
	C[(A_start_index+1)*C_width + B_start_index+0] = c_1_0;
	C[(A_start_index+1)*C_width + B_start_index+1] = c_1_1;
	C[(A_start_index+1)*C_width + B_start_index+2] = c_1_2;
	C[(A_start_index+2)*C_width + B_start_index+0] = c_2_0;
	C[(A_start_index+2)*C_width + B_start_index+1] = c_2_1;
	C[(A_start_index+2)*C_width + B_start_index+2] = c_2_2;


}
