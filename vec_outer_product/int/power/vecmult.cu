#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include "vecmultKernel.h"

#include "high_resolution_power.h"

// Variables for host and device vectors.
int* h_A; 
int* h_B; 
int* h_C; 
int* d_A; 
int* d_B; 
int* d_C; 

    int ValuesPerThread; // number of values per thread
	int k; // no. of repeatitions
	int gridWidth = 60;
	int blockWidth = 1;
// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

void call_gpu_function() {
    dim3 dimGrid(gridWidth);                    
    dim3 dimBlock(blockWidth);                 

    MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);

}

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
    int N; //Vector size

	// Parse arguments.
    if(argc != 5){
     printf("Usage: %s ValuesPerThread Iterations\n", argv[0]);
     printf("ValuesPerThread is the number of values added by each thread.\n");
     printf("Total vector size is 128 * 60 * this value.\n");
     printf("Iterations is the number of repeatitions done by each thread.\n");
     exit(0);
    } else {
      sscanf(argv[1], "%d", &ValuesPerThread);
      sscanf(argv[2], "%d", &k);
      sscanf(argv[3], "%d", &gridWidth);
      sscanf(argv[4], "%d", &blockWidth);
    }      

	int size_A = blockWidth * ValuesPerThread;
	int size_B = gridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);

    // Determine the number of threads .
    // N is the total number of values to be in a vector
//    N = ValuesPerThread * gridWidth * blockWidth;
//    printf("Total vector size: %d : Iterations: %d\n", N, k); 
    // size_t is the total number of bytes for a vector.
//    size_t size = N * sizeof(int);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.

    // Allocate input vectors h_A and h_B in host memory
    h_A = (int*)malloc(size_A*sizeof(int));
    if (h_A == 0) Cleanup(false);
    h_B = (int*)malloc(size_B*sizeof(int));
    if (h_B == 0) Cleanup(false);
    h_C = (int*)malloc(size_A*sizeof(int)*size_B*sizeof(int));
    if (h_C == 0) Cleanup(false);
	
//	printf("1 \n");

    // Allocate vectors in device memory.
    hipError_t error;
    error = hipMalloc((void**)&d_A, size_A*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
	//printf("1.1\n");
    error = hipMalloc((void**)&d_B, size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);
//	printf("1.2\n");
    error = hipMalloc((void**)&d_C, size_A*sizeof(int)*size_B*sizeof(int));
    if (error != hipSuccess) Cleanup(false);

//	printf("2\n");
    // Initialize host vectors h_A and h_B
    int i, j;
    for(i=0; i <size_A; ++i){
     h_A[i] = (int)i;
    }
    for(i=0; i <size_B; ++i){
     h_B[i] = (int)(N-i);   
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    error = hipMemcpy(d_A, h_A, size_A*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);
    error = hipMemcpy(d_B, h_B, size_B*sizeof(int), hipMemcpyHostToDevice);
    if (error != hipSuccess) Cleanup(false);

//	printf("3\n");
    // Warm up
  //  MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);
    call_gpu_function();
    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("W: %s\n", hipGetErrorString(error));
		Cleanup(false);
	}
    hipDeviceSynchronize();

    // Invoke kernel
    //MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, ValuesPerThread, k);

    call_gpu_function();

    error = hipGetLastError();
    if (error != hipSuccess) {
		printf("%s\n", hipGetErrorString(error));
		Cleanup(false);
	}

//	printf("5\n");
	// Compute elapsed time 
    hipDeviceSynchronize();

    long long exec_time_ns = get_exec_time_in_nanoseconds(call_gpu_function);
    double time = exec_time_ns/1e6; //ms
    //float time = cutGetTimerValue(timer);

	// Compute floating point operations per second.
    //double nFlops = size_A*size_B*k*2 + 2*k*ValuesPerThread*gridWidth*blockWidth;
    double nFlops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)4*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth;
    //double nFlops = (double)size_A*(double)size_B*(double)k*(double)2;
    double nFlopsPerSec = 1e3*nFlops/time;
    double nGFlopsPerSec = nFlopsPerSec*1e-9;
	//printf("%f :: %f", (double)size_A*(double)size_B*(double)k*(double)2, (double)2*(double)k*(double)ValuesPerThread*(double)gridWidth*(double)blockWidth);
	// Compute transfer rates.
    double nBytes = size_A*sizeof(int) + size_B*sizeof(int) + size_A*sizeof(int)*size_B*sizeof(int); // 2N words in, N*N word out
    double nBytesPerSec = 1e3*nBytes/time;
    double nGBytesPerSec = nBytesPerSec*1e-9;

	// Report timing data.
    printf( "Time: %f (ms), GOPS: %f, GBytesS: %f\n", 
             time, nGFlopsPerSec, nGBytesPerSec);
    
    // Copy result from device memory to host memory
    error = hipMemcpy(h_C, d_C, size_A*sizeof(int)*size_B*sizeof(int), hipMemcpyDeviceToHost);
    if (error != hipSuccess) Cleanup(false);

    // Verify & report result
    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
			int val = h_C[i*size_B+j];
			if (abs(val - h_A[i]*h_B[j]) > 0) {
				printf("Result error: i=%d, j=%d, expected %d, got %d\n", i, j, h_A[i]*h_B[j], val);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    }
    printf("Test %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	// Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    hipError_t error;
        
    // Free device vectors
    if (d_A)
        hipFree(d_A);
    if (d_B)
        hipFree(d_B);
    if (d_C)
        hipFree(d_C);

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    error = hipDeviceReset();
   checkCUDAError("tmp"); 
    if (!noError || error != hipSuccess)
        printf("error: %s cuda malloc or cuda thread exit failed \n", hipGetErrorString(hipGetLastError()));
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}

void checkCUDAError(const char *msg)
{
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) 
    {
		printf("Error");
      fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString(err) );
      exit(-1);
    }                         
}


