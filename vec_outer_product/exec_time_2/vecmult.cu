#include "hip/hip_runtime.h"
// Includes
#include <stdio.h>
#include "vecmultKernel.h"

#include "high_resolution_power.h"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                    \
    hipError_t _m_cudaStat = value;                    \
    if (_m_cudaStat != hipSuccess) {                   \
      fprintf(stderr, "Error: %s at line %d in file %s\n",        \
          hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);   \
          exit(1);        \
    } }



// Variables for host and device vectors.
float* h_A; 
float* h_B; 
float* h_C; 
float* d_A; 
float* d_B; 
float* d_C; 

float* h_C_cpu; 
//int GridWidth;
//int BlockWidth;
//int repetitions;
// Utility Functions
void Cleanup(bool);
void checkCUDAError(const char *msg);

void cuda_function_call() {
  dim3 dimGrid(GridWidth);                    
  dim3 dimBlock(BlockWidth);                 
  MultiplyVectors<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
  CUDA_CHECK_RETURN(hipGetLastError()); 
}

// Host code performs setup and calls the kernel.
int main(int argc, char** argv)
{
  int N; //Vector size

  int i, j, t;
	// Parse arguments.
    if(argc != 1){
     printf("Usage: %s GridWidth BlockWidth\n", argv[0]);
     printf("GridWidth - number of thread blocks.\n");
     printf("BlockWidth - number of threads per thread block\n");
    // printf("Repetitions - number of repetitions of the computations\n");
     exit(0);
    } else {
//      sscanf(argv[1], "%d", &ValuesPerThread);
//      sscanf(argv[2], "%d", &k);
  //    sscanf(argv[1], "%d", &GridWidth);
//      sscanf(argv[2], "%d", &BlockWidth);
 //     sscanf(argv[3], "%d", &repetitions);
    }      
  //int k = repetitions;
	int size_A = BlockWidth * ValuesPerThread;
	int size_B = GridWidth * ValuesPerThread;

	printf("Size of A: %d, Size of B: %d\n", size_A, size_B);

    // Determine the number of threads .
    // N is the total number of values to be in a vector
//    N = ValuesPerThread * GridWidth * BlockWidth;
//    printf("Total vector size: %d : Iterations: %d\n", N, k); 
    // size_t is the total number of bytes for a vector.
//    size_t size = N * sizeof(float);

    // Tell CUDA how big to make the grid and thread blocks.
    // Since this is a vector addition problem,
    // grid and thread block are both one-dimensional.
    dim3 dimGrid(GridWidth);                    
    dim3 dimBlock(BlockWidth);                 

    // Allocate input vectors h_A and h_B in host memory
    h_A = (float*)malloc(size_A*sizeof(float));
    if (h_A == 0) Cleanup(false);
    h_B = (float*)malloc(size_B*sizeof(float));
    if (h_B == 0) Cleanup(false);
    h_C = (float*)malloc(size_A*size_B*sizeof(float));
    if (h_C == 0) Cleanup(false);
    
    h_C_cpu = (float*)malloc(size_A*size_B*sizeof(float));
    if (h_C_cpu == 0) Cleanup(false);
	
//	printf("1 \n");

    // Allocate vectors in device memory.
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_A, size_A*sizeof(float)));
	//printf("1.1\n");
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_B, size_B*sizeof(float)));
//	printf("1.2\n");
    CUDA_CHECK_RETURN(hipMalloc((void**)&d_C, size_A*size_B*sizeof(float)));

//	printf("2\n");
    // Initialize host vectors h_A and h_B
    for(i=0; i <size_A; ++i){
     h_A[i] = (float)i;
    }
    for(i=0; i <size_B; ++i){
     h_B[i] = (float)(size_B-i);   
    }

    // Copy host vectors h_A and h_B to device vectores d_A and d_B
    CUDA_CHECK_RETURN(hipMemcpy(d_A, h_A, size_A*sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(d_B, h_B, size_B*sizeof(float), hipMemcpyHostToDevice));


//	printf("4\n");
    
//	printf("5\n");
	// Compute elapsed time 
    long long exec_time = 0;
    exec_time = get_exec_time_in_nanoseconds(cuda_function_call);
    double time = exec_time/1e9; //s

	// Compute floating point operations per second.
    //double nFlops = size_A*size_B*k*2 + 2*k*ValuesPerThread*GridWidth*BlockWidth;
    //double nFlops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)2*(double)k*(double)ValuesPerThread*(double)GridWidth*(double)BlockWidth;
    double nFlops = (double)size_A*(double)size_B*(double)k*(double)2 + (double)2*(double)k*(double)ValuesPerThread*(double)GridWidth*(double)BlockWidth*(double)2;
    //float nFlopsPerSec = 1e3*nFlops/time;
    //float nGFlopsPerSec = nFlopsPerSec*1e-9;

    double nGFlopsPerSec = nFlops/exec_time;

	//printf("%f :: %f", (double)size_A*(double)size_B*(double)k*(double)2, (double)2*(double)k*(double)ValuesPerThread*(double)GridWidth*(double)BlockWidth);
	// Compute transfer rates.
    double nBytes = size_A*sizeof(float) + size_B*sizeof(float) + size_A*size_B*sizeof(float); // 2N words in, N*N word out
    //float nBytesPerSec = 1e3*nBytes/time;
    //float nGBytesPerSec = nBytesPerSec*1e-9;
    double nGBytesPerSec = nBytes/exec_time;

	// Report timing data.
    printf( "%d %d %d %d Time: %f (ms), GFLOPS: %f GBytesS: %f\n", 
             GridWidth, BlockWidth, k, ValuesPerThread, time*1e3, nGFlopsPerSec, nGBytesPerSec);
     
    // Copy result from device memory to host memory
    CUDA_CHECK_RETURN(hipMemcpy(h_C, d_C, size_A*size_B*sizeof(float), hipMemcpyDeviceToHost));

    // Verify & report result
    memset(h_C_cpu, 0, size_A*size_B*sizeof(float));

    for (t = 0; t < k; t++) {
      if (t > 0) {
        for (i = 0; i < size_A; i++) {
          h_A[i] = h_A[i]*1.1f + 1.7f;
        }
        for (j = 0; j < size_B; j++) {
          h_B[j] = h_B[j]*1.1f + 1.7f;
        }
      }
      for (i = 0; i < size_A; i++) {
        for (j = 0; j < size_B; j++) {
          h_C_cpu[i*size_B + j] += h_A[i]*h_B[j];
        }
      }
      
    }


    printf("Result [%d,%d] expected %f*%f=%.5f, got %.5f, error: %e\n", 0,0, h_A[0], h_B[0], h_C_cpu[0], h_C[0], h_C[0] - h_C_cpu[0]);
/*
    for(i=0; i <size_A; ++i){
     printf("%.1f ", h_A[i]);
    }
     printf("\n");
    for(i=0; i <size_B; ++i){
     printf("%.1f ", h_B[i]);
    }
     printf("\n");

    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
        printf("%.1f,%.1f ", h_C[i*size_B+j], h_C_cpu[i*size_B+j]);
      }
      printf(":");
    }
*/
    for (i = 0; i < size_A; ++i) {
    	for (j = 0; j < size_B; ++j) {
			float val = h_C[i*size_B+j];
			if (fabs(val - h_C_cpu[i*size_B+j] ) > 1e0) {
				printf("Result error: i=%d, j=%d, expected %.5f, got %.5f, error: %e\n", i, j, h_C_cpu[i*size_B+j], val, val - h_C_cpu[i*size_B+j]);
				break;
			}
		}
		if (j != size_B) {
			break;
		}
    }
    printf("\nTest %s \n", (i == size_A && j == size_B) ? "PASSED" : "FAILED");

	// Clean up and exit.
    Cleanup(true);
}

void Cleanup(bool noError) {  // simplified version from CUDA SDK
    // Free device vectors
    if (d_A)
        CUDA_CHECK_RETURN(hipFree(d_A));
    if (d_B)
        CUDA_CHECK_RETURN(hipFree(d_B));
    if (d_C)
        CUDA_CHECK_RETURN(hipFree(d_C));

    // Free host memory
    if (h_A)
        free(h_A);
    if (h_B)
        free(h_B);
    if (h_C)
        free(h_C);
        
    
    fflush( stdout);
    fflush( stderr);

    exit(0);
}


