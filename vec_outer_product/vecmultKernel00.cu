#include "hip/hip_runtime.h"
/** size of A = 4
    size of B = 4
    gridDim = 2
    blockDim = 2
    k= 2
    x = 2
**/

#include "vecmultKernel.h"
__global__ void MultiplyVectors(const float* A, const float* B, float* C)
{
	int B_start_index = (blockIdx.x)*k;
	int A_start_index = (threadIdx.x)*k;
	int C_width = k*gridDim.x;


	int t;
	float c_0_0, c_0_1, c_1_0, c_1_1;
	float a_0, a_1;
	float b_0, b_1;


	a_0 = A[A_start_index+0];
	a_1 = A[A_start_index+1];


	b_0 = B[B_start_index+0];
	b_1 = B[B_start_index+1];


	c_0_0 = 0;
	c_0_1 = 0;
	c_1_0 = 0;
	c_1_1 = 0;


	for (t = 0; t < 2; t++) {
		c_0_0 += a_0*b_0;
		c_0_1 += a_0*b_1;
		c_1_0 += a_1*b_0;
		c_1_1 += a_1*b_1;


/*
		a_0 += 10;
		a_1 += 10;


		b_0 += 10;
		b_1 += 10;


*/
	}


	C[(A_start_index+0)*C_width + B_start_index+0] = c_0_0;
	C[(A_start_index+0)*C_width + B_start_index+1] = c_0_1;
	C[(A_start_index+1)*C_width + B_start_index+0] = c_1_0;
	C[(A_start_index+1)*C_width + B_start_index+1] = c_1_1;


}
